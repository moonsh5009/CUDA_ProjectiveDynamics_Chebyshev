#include "hip/hip_runtime.h"
#include "Mesh.h"
#include "../include/CUDA_Custom/DeviceManager.cuh"

void Mesh::loadObj(const char* filename, REAL3 center, REAL scale) {
	CUDA_CHECK(hipDeviceSynchronize());
	ctimer timer = CNOW;

	_fs.clear();
	_ns.clear();

	bool flag = true;
	ifstream fin;
	fin.open(filename);
	if (fin.is_open()) {
		while (!fin.eof()) {
			string head;
			fin >> head;
			if (head.length() > 1)
				continue;
			if (head[0] == 'v') {
				REAL3 x;
				fin >> x.x >> x.y >> x.z;
				_ns.push_back(x.x);
				_ns.push_back(x.y);
				_ns.push_back(x.z);
				if (flag) {
					_aabb._min = _aabb._max = x;
					flag = false;
				}
				else addAABB(_aabb, x);
			}
			else if (head[0] == 'f') {
				uint3 x;
				fin >> x.x >> x.y >> x.z;
				_fs.push_back(x.x - 1u);
				_fs.push_back(x.y - 1u);
				_fs.push_back(x.z - 1u);
			}
		}
		fin.close();
	}
	if (_ns.empty() || _fs.empty()) {
		printf("Error : Mesh_init : Object Load Error\n");
		exit(1);
		return;
	}
	_numFaces = _fs.size() / 3u;
	_numVertices = _ns.size() / 3u;
	moveCenter(center, scale);
	buildAdjacency();
	computeNormal();

	CUDA_CHECK(hipDeviceSynchronize());
	printf("Num of Faces: %d, Num of Vertices: %d, %f ms\n", _numFaces, _numVertices, (CNOW - timer) / 10000.);
}
void Mesh::loadObj(const char* filename) {
	CUDA_CHECK(hipDeviceSynchronize());
	ctimer timer = CNOW;

	_fs.clear();
	_ns.clear();

	bool flag = true;
	ifstream fin;
	fin.open(filename);
	if (fin.is_open()) {
		while (!fin.eof()) {
			string head;
			fin >> head;
			if (head.length() > 1)
				continue;
			if (head[0] == 'v') {
				REAL3 x;
				fin >> x.x >> x.y >> x.z;
				_ns.push_back(x.x);
				_ns.push_back(x.y);
				_ns.push_back(x.z);
				if (flag) {
					_aabb._min = _aabb._max = x;
					flag = false;
				}
				else addAABB(_aabb, x);
			}
			else if (head[0] == 'f') {
				uint3 x;
				fin >> x.x >> x.y >> x.z;
				_fs.push_back(x.x - 1u);
				_fs.push_back(x.y - 1u);
				_fs.push_back(x.z - 1u);
			}
		}
		fin.close();
	}
	if (_ns.empty() || _fs.empty()) {
		printf("Error : Mesh_init : Object Load Error\n");
		exit(1);
		return;
	}
	_numFaces = _fs.size() / 3u;
	_numVertices = _ns.size() / 3u;
	buildAdjacency();
	computeNormal();

	CUDA_CHECK(hipDeviceSynchronize());
	printf("Num of Faces: %d, Num of Vertices: %d, %f ms\n", _numFaces, _numVertices, (CNOW - timer) / 10000.);
}

void Mesh::moveCenter(REAL3 center, REAL scale) {
	REAL3 size = _aabb._max - _aabb._min;
	REAL max_length = size.x;
	if (max_length < size.y)
		max_length = size.y;
	if (max_length < size.z)
		max_length = size.z;
	max_length = 2.0 * scale / max_length;

	REAL3 prevCenter = (_aabb._min + _aabb._max) * (REAL)0.5;

	bool flag = false;
	uint vlen = _ns.size();
	for (uint i = 0u; i < vlen; i += 3u) {
		REAL3 pos = make_REAL3(_ns[i], _ns[i + 1u], _ns[i + 2u]);
		REAL3 grad = pos - prevCenter;
		grad *= max_length;
		pos = center + grad;
		_ns[i] = pos.x;
		_ns[i + 1u] = pos.y;
		_ns[i + 2u] = pos.z;
		if (flag) addAABB(_aabb, pos);
		else {
			_aabb._min = _aabb._max = pos;
			flag = true;
		}
	}
}
void Mesh::buildAdjacency(void)
{
	vector<set<uint>> nbFs(_numVertices);
	vector<set<uint>> nbNs(_numVertices);

	for (uint i = 0u; i < _numFaces; i++) {
		uint ino = i * 3u;
		uint ino0 = _fs[ino + 0u];
		uint ino1 = _fs[ino + 1u];
		uint ino2 = _fs[ino + 2u];
		nbFs[ino0].insert(i);
		nbFs[ino1].insert(i);
		nbFs[ino2].insert(i);
		nbNs[ino0].insert(ino1);
		nbNs[ino0].insert(ino2);
		nbNs[ino1].insert(ino2);
		nbNs[ino1].insert(ino0);
		nbNs[ino2].insert(ino0);
		nbNs[ino2].insert(ino1);
	}

	vector<uint> ses;
	vector<uint> bes;
	for (uint i = 0u; i < _numVertices; i++) {
		for (auto inbv : nbNs[i]) {
			if (i < inbv) {
				ses.push_back(i);
				ses.push_back(inbv);
			}
		}
	}

	uint ns[2];
	for (uint i = 0u; i < ses.size(); i += 2) {
		uint ino0 = ses[i + 0u];
		uint ino1 = ses[i + 1u];
		uint num = 0u;
		for (auto inbf0 : nbFs[ino0]) {
			for (auto inbf1 : nbFs[ino1]) {
				if (inbf0 == inbf1) {
					uint iface = inbf0 * 3u;
					for (uint j = 0u; j < 3u; j++) {
						uint ino = _fs[iface + j];
						if (ino != ino0 && ino != ino1)
							ns[num++] = ino;
					}
					break;
				}
			}
			if (num == 2u) break;
		}
		if (num == 2u) {
			num = ns[0] > ns[1];
			bes.push_back(ns[num]);
			bes.push_back(ns[num ^ 1u]);
		}
	}

	_nbFs.init(nbFs);
	_nbNs.init(nbNs);

	DPrefixArray<uint> d_ses;
	DPrefixArray<uint> d_bes;
	d_ses._array = ses;
	d_bes._array = bes;

	thrust::sort(thrust::device_ptr<uint2>((uint2*)d_ses._array.begin()),
		thrust::device_ptr<uint2>((uint2*)d_ses._array.end()), uint2_CMP());
	thrust::sort(thrust::device_ptr<uint2>((uint2*)d_bes._array.begin()),
		thrust::device_ptr<uint2>((uint2*)d_bes._array.end()), uint2_CMP());
	d_ses._index.resize(_numVertices + 1u);
	d_bes._index.resize(_numVertices + 1u);

	uint numEdges = d_ses._array.size() >> 1u;
	reorderIdsUint2_kernel << < divup(numEdges, MAX_BLOCKSIZE), MAX_BLOCKSIZE, (MAX_BLOCKSIZE + 1u) * sizeof(uint) >> > (
		(uint2*)d_ses._array(), d_ses._index(), numEdges, d_ses._index.size());
	CUDA_CHECK(hipPeekAtLastError());
	numEdges = d_bes._array.size() >> 1u;
	reorderIdsUint2_kernel << < divup(numEdges, MAX_BLOCKSIZE), MAX_BLOCKSIZE, (MAX_BLOCKSIZE + 1u) * sizeof(uint) >> > (
		(uint2*)d_bes._array(), d_bes._index(), numEdges, d_bes._index.size());
	CUDA_CHECK(hipPeekAtLastError());

	d_ses.copyToHost(_ses);
	d_bes.copyToHost(_bes);
}
void Mesh::computeNormal(void)
{
	uint numEdges = _ses.arraySize();
	_fnorms.resize(_numFaces * 3u);
	_vnorms.clear();
	_vnorms.resize(_numVertices * 3u, 0.0);

	for (uint i = 0u; i < _numFaces; i++) {
		uint ino = i * 3u;
		uint ino0 = _fs[ino + 0u] * 3u;
		uint ino1 = _fs[ino + 1u] * 3u;
		uint ino2 = _fs[ino + 2u] * 3u;
		REAL3 a = make_REAL3(_ns[ino0 + 0u], _ns[ino0 + 1u], _ns[ino0 + 2u]);
		REAL3 b = make_REAL3(_ns[ino1 + 0u], _ns[ino1 + 1u], _ns[ino1 + 2u]);
		REAL3 c = make_REAL3(_ns[ino2 + 0u], _ns[ino2 + 1u], _ns[ino2 + 2u]);
		REAL3 norm = Cross(a - b, a - c);
		Normalize(norm);
		_fnorms[ino + 0u] = norm.x;
		_fnorms[ino + 1u] = norm.y;
		_fnorms[ino + 2u] = norm.z;
		REAL radian = AngleBetweenVectors(a - b, a - c);
		_vnorms[ino0 + 0u] += norm.x * radian;
		_vnorms[ino0 + 1u] += norm.y * radian;
		_vnorms[ino0 + 2u] += norm.z * radian;
		radian = AngleBetweenVectors(b - a, b - c);
		_vnorms[ino1 + 0u] += norm.x * radian;
		_vnorms[ino1 + 1u] += norm.y * radian;
		_vnorms[ino1 + 2u] += norm.z * radian;
		radian = AngleBetweenVectors(c - a, c - b);
		_vnorms[ino2 + 0u] += norm.x * radian;
		_vnorms[ino2 + 1u] += norm.y * radian;
		_vnorms[ino2 + 2u] += norm.z * radian;
	}

	for (uint i = 0u; i < _numVertices; i++) {
		uint ino = i * 3u;
		REAL3 norm = make_REAL3(_vnorms[ino + 0u], _vnorms[ino + 1u], _vnorms[ino + 2u]);
		Normalize(norm);
		_vnorms[ino + 0u] = norm.x;
		_vnorms[ino + 1u] = norm.y;
		_vnorms[ino + 2u] = norm.z;
	}
}
void Mesh::rotate(REAL3 degree)
{
	degree.x *= M_PI * 0.00555555555555555555555555555556;
	degree.y *= M_PI * 0.00555555555555555555555555555556;
	degree.z *= M_PI * 0.00555555555555555555555555555556;

	REAL cx = cos(degree.x);
	REAL sx = sin(degree.x);
	REAL cy = cos(degree.y);
	REAL sy = -sin(degree.y);
	REAL cz = cos(degree.z);
	REAL sz = sin(degree.z);
	REAL3 center = (_aabb._min + _aabb._max) * (REAL)0.5;

	REAL3 n, pn;
	for (uint i = 0u; i < _ns.size(); i+=3u) {
		n.x = _ns[i + 0u];
		n.y = _ns[i + 1u];
		n.z = _ns[i + 2u];
		n -= center;

		pn.x = n.x * cz * cy + n.y * (cz * sy * sx - sz * cx) + n.z * (cz * sy * cx + sz * sx);
		pn.y = n.x * sz * cy + n.y * (sz * sy * sx + cz * cx) + n.z * (sz * sy * cx - cz * sx);
		pn.z = n.x * -sy + n.y * cy * sx + n.z * cy * cx;

		pn += center;

		_ns[i + 0u] = pn.x;
		_ns[i + 1u] = pn.y;
		_ns[i + 2u] = pn.z;
	}

	computeNormal();
}