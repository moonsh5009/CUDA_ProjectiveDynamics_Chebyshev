#include "BVH.cuh"

struct TriInfo_CMP
{
	__host__ __device__
		bool operator()(const TriInfo& a, const TriInfo& b) {
		if (a._id != b._id)
			return a._id < b._id;
		return a._pos < b._pos;
	}
};

void BVH::build(Dvector<uint>& fs, Dvector<REAL>& ns) {
	CUDA_CHECK(hipDeviceSynchronize());
	ctimer timer = CNOW;

	uint numFaces = fs.size() / 3u;
	Dvector<TriInfo> infos(numFaces);

	initBVHTreeDevice(numFaces);
	initBVHInfo_kernel << < divup(numFaces, BLOCKSIZE), BLOCKSIZE >> > (
		fs(), ns(), infos(), param());
	CUDA_CHECK(hipPeekAtLastError());
	InitMinMaxKernel << < divup(numFaces, MAX_BLOCKSIZE), MAX_BLOCKSIZE >> > (
		param());
	CUDA_CHECK(hipPeekAtLastError());
	updateBVHInfo_kernel << < divup(numFaces, MAX_BLOCKSIZE), MAX_BLOCKSIZE >> > (
		infos(), param());
	CUDA_CHECK(hipPeekAtLastError());
	thrust::sort(thrust::device_ptr<TriInfo>(infos.begin()),
		thrust::device_ptr<TriInfo>(infos.end()), TriInfo_CMP());

	for (uint level = 0u; level < _maxLevel - 2u; level++) {
		uint size = 1u << level;
		updateMinMax_kernel << < divup(size, BLOCKSIZE), BLOCKSIZE >> > (
			param(), level, size);
		CUDA_CHECK(hipPeekAtLastError());
		subdivBVH_kernel << < divup(numFaces, BLOCKSIZE), BLOCKSIZE >> > (
			infos(), param());
		CUDA_CHECK(hipPeekAtLastError());
		updateBVHInfo_kernel << < divup(numFaces, MAX_BLOCKSIZE), MAX_BLOCKSIZE >> > (
			infos(), param());
		CUDA_CHECK(hipPeekAtLastError());
		thrust::sort(thrust::device_ptr<TriInfo>(infos.begin()),
			thrust::device_ptr<TriInfo>(infos.end()), TriInfo_CMP());
		/*{
			vector<TriInfo> hinfo;
			infos.copyToHost(hinfo);
			printf("%d: ", level);
			for (auto t : hinfo) {
				printf("%d ", t._id);
			}
			printf("\n");
		}*/
	}
	buildBVH_kernel << < divup(_size, MAX_BLOCKSIZE), MAX_BLOCKSIZE >> > (
		infos(), param());
	CUDA_CHECK(hipPeekAtLastError());

		/*vector<TriInfo> hinfo;
		infos.copyToHost(hinfo);
		for (auto t : hinfo) {
			printf("%d\n", t._id);
		}
		printf("\n");*/

	_test = 0u;

	CUDA_CHECK(hipDeviceSynchronize());
	printf("BVH Build: %lf msec\n", (CNOW - timer) / 10000.0);
}
void BVH::refit(ObjParam obj, REAL delta, const REAL dt, bool isCCD) {
	if (!_size)
		return;
	//CUDA_CHECK(hipDeviceSynchronize());
	//ctimer timer = CNOW;

	//if (!isCCD)	delta *= 0.5;

	uint currLevel = _maxLevel - 1u;
	int num = 1u << (currLevel--);
	RefitLeafBVHKernel << < divup(num, REFIT_BLOCKSIZE), REFIT_BLOCKSIZE >> >
		(obj, param(), num, delta, dt, isCCD);
	CUDA_CHECK(hipPeekAtLastError());

	while (currLevel > 10u) {
		num >>= 1; currLevel--;
		RefitBVHKernel << < divup(num, REFIT_BLOCKSIZE), REFIT_BLOCKSIZE >> >
			(param(), num);
		CUDA_CHECK(hipPeekAtLastError());
	}
	RefitNodeBVHKernel << < 1u, MAX_BLOCKSIZE >> >
		(param(), currLevel);
	CUDA_CHECK(hipPeekAtLastError());

	//CUDA_CHECK(hipDeviceSynchronize());
	//printf("BVH Refit: %lf msec\n", (CNOW - timer) / 10000.0);
}
void BVH::refit(uint* fs, REAL* ns, REAL delta) {
	if (!_size)
		return;
	//CUDA_CHECK(hipDeviceSynchronize());
	//ctimer timer = CNOW;

	//if (!isCCD)	delta *= 0.5;

	uint currLevel = _maxLevel - 1u;
	int num = 1u << (currLevel--);
	RefitLeafBVHKernel << < divup(num, REFIT_BLOCKSIZE), REFIT_BLOCKSIZE >> >
		(fs, ns, param(), num, delta);
	CUDA_CHECK(hipPeekAtLastError());

	while (currLevel > 10u) {
		num >>= 1; currLevel--;
		RefitBVHKernel << < divup(num, REFIT_BLOCKSIZE), REFIT_BLOCKSIZE >> >
			(param(), num);
		CUDA_CHECK(hipPeekAtLastError());
	}
	RefitNodeBVHKernel << < 1u, MAX_BLOCKSIZE >> >
		(param(), currLevel);
	CUDA_CHECK(hipPeekAtLastError());

	//CUDA_CHECK(hipDeviceSynchronize());
	//printf("BVH Refit: %lf msec\n", (CNOW - timer) / 10000.0);
}
void BVH::refit(uint* fs, REAL* ns, REAL* vs, REAL delta, REAL dt) {
	if (!_size)
		return;
	//CUDA_CHECK(hipDeviceSynchronize());
	//ctimer timer = CNOW;

	uint currLevel = _maxLevel - 1u;
	int num = 1u << (currLevel--);
	RefitLeafBVHKernel << < divup(num, REFIT_BLOCKSIZE), REFIT_BLOCKSIZE >> >
		(fs, ns, vs, param(), num, delta, dt);
	CUDA_CHECK(hipPeekAtLastError());

	while (currLevel > 10u) {
		num >>= 1; currLevel--;
		RefitBVHKernel << < divup(num, REFIT_BLOCKSIZE), REFIT_BLOCKSIZE >> >
			(param(), num);
		CUDA_CHECK(hipPeekAtLastError());
	}
	RefitNodeBVHKernel << < 1u, MAX_BLOCKSIZE >> >
		(param(), currLevel);
	CUDA_CHECK(hipPeekAtLastError());

	//CUDA_CHECK(hipDeviceSynchronize());
	//printf("BVH Refit: %lf msec\n", (CNOW - timer) / 10000.0);
}
void BVH::draw(const AABB& aabb) {
	glDisable(GL_LIGHTING);
	glPushMatrix();
	glLineWidth(1.0f);
	//glColor3d(0.6, 0.6, 0.6);
	glColor3d(1.0, 0.0, 0.0);
	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
	glBegin(GL_LINES);
	glVertex3d(aabb._min.x, aabb._min.y, aabb._min.z);
	glVertex3d(aabb._min.x, aabb._min.y, aabb._max.z);
	glVertex3d(aabb._min.x, aabb._max.y, aabb._min.z);
	glVertex3d(aabb._min.x, aabb._max.y, aabb._max.z);
	glVertex3d(aabb._max.x, aabb._min.y, aabb._min.z);
	glVertex3d(aabb._max.x, aabb._min.y, aabb._max.z);
	glVertex3d(aabb._max.x, aabb._max.y, aabb._min.z);
	glVertex3d(aabb._max.x, aabb._max.y, aabb._max.z);
	glEnd();
	glTranslated(0, 0, aabb._min.z);
	glRectd(aabb._min.x, aabb._min.y, aabb._max.x, aabb._max.y);
	glTranslated(0, 0, aabb._max.z - aabb._min.z);
	glRectd(aabb._min.x, aabb._min.y, aabb._max.x, aabb._max.y);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	glPopMatrix();
	glEnable(GL_LIGHTING);
}
void BVH::draw(void) {
	if (!_size)
		return;
#ifndef CHECK_DETECTION
	vector<REAL> mins[3];
	vector<REAL> maxs[3];
	vector<uint> levels;
	for (uint i = 0u; i < 3u; i++) {
		_mins[i].copyToHost(mins[i]);
		_maxs[i].copyToHost(maxs[i]);
	}
	_levels.copyToHost(levels);
	//uint testlevel = levels[_test];
	//uint testpath = _test - (1u << levels[_test]) + 1;
	for (uint i = 0u; i < _size; i++) {
		AABB aabb;
		aabb._min = make_REAL3(mins[0][i], mins[1][i], mins[2][i]);
		aabb._max = make_REAL3(maxs[0][i], maxs[1][i], maxs[2][i]);
		draw(aabb);
		/*if (levels[i] > testlevel && i > _size - _numFaces) {
			uint path = i - (1u << levels[i]) + 1;
			path >>= (levels[i] - testlevel);
			if (path == testpath) {
				draw(aabb);
			}
		}
		if (i == _test) {
			draw(aabb);
		}*/
	}
#else
	vector<REAL> mins[3];
	vector<REAL> maxs[3];
	vector<uint> levels;
	vector<uint> isDetecteds;
	vector<uint> faces;
	for (uint i = 0u; i < 3u; i++) {
		_mins[i].copyToHost(mins[i]);
		_maxs[i].copyToHost(maxs[i]);
	}
	_levels.copyToHost(levels);
	_isDetecteds.copyToHost(isDetecteds);
	_faces.copyToHost(faces);

	uint ileaf = _size - _numFaces;
	for (uint i = 0u; i < _numFaces; i++) {
		AABB aabb;
		uint face = faces[i];
		if (isDetecteds[face]) {
			aabb._min = make_REAL3(mins[0][i + ileaf], mins[1][i + ileaf], mins[2][i + ileaf]);
			aabb._max = make_REAL3(maxs[0][i + ileaf], maxs[1][i + ileaf], maxs[2][i + ileaf]);
			draw(aabb);
		}
	}
#endif
}